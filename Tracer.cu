#include "hip/hip_runtime.h"
#include <host_defines.h>
#include "Tracer.h"
#include "Render.h"
#include "Object/Light/PointLightRay3.h"

__global__ void recursive_cuda(Ray3 * d_ray, int * d_times, double * d_distance){
    int h = blockIdx.x;
    int w = threadIdx.x;
}

void Tracer::Scan_cuda(int times) {

}

Color3 Tracer::recursive_cuda(Ray3* ray, int times, double decay, double& distance) {
    times--;
    Color3 res = Color3(0,0,0);
//    Intersection sight_hit = ray_with_objs(ray,obj_list);
//    if(sight_hit.isHit){
//        res = sight_hit.object->material->ambient;
//        for(LightSource* temp_light:light_src_list) {
//            LightRay3* temp_light_ray = temp_light->traceLight(sight_hit.position);//+1e-3*sight_hit.normal);
//            Intersection light_hit = ray_with_objs(*temp_light_ray, obj_list);
//            if (!light_hit.isHit || light_hit.object == temp_light) {
//                Color3 color = sight_hit.object->material->sample(ray, temp_light_ray, sight_hit.distance, sight_hit.normal);
//                res += color*sight_hit.object->material->reflectivity;
//            }
//            delete temp_light_ray;
//        }
//        if(times>0){
//            if(sight_hit.object->material->trace_reflectivity>0) {
//                Ray3 reflect(sight_hit.position+sight_hit.normal*1e-3, ray.direction - 2 * ray.direction.project(sight_hit.normal));
//                Color3 ref = recursive(reflect, times, sight_hit.object->material->reflectivity,distance);
//                PointLightRay3 light_reflect(reflect,ref);
//                //LightRay3 light_reflect(reflect,ref);
//                Color3 color = sight_hit.object->material->sample(ray, &light_reflect, distance, sight_hit.normal);
//                res += color*sight_hit.object->material->trace_reflectivity;
//            }
////            if(sight_hit.object->material->refractivity>0){// to be done, transparent material
////                Vector3 proj = ray.direction.project(sight_hit.normal);
////                Vector3 parallel = ray.direction + proj;
////                double sin_theta1 = parallel.getLength();
////                double sin_theta2 = sin_theta1/sight_hit.object->material->n;
////                double tan_theta2 = sin_theta2/sqrt(1-sin_theta2*sin_theta2);
////                Ray3 refract(sight_hit.position,sight_hit.normal+parallel.normalize()*tan_theta2);
////                recursive(refract, times, sight_hit.object->material->reflectivity);
////            }
//        }
//    }
//    distance = sight_hit.distance;
    return res;
}